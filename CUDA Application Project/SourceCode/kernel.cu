#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

#include "gpuErrchk.h"

/*
 GPU device function that performs the check to see if the found token is a non letter or not 
*/
__device__ bool is_non_letter(char c) {
     
    return (c < 'a' || c > 'z');
}

/*
 GPU device function that performs the compare token check by comparing the current index
 Threads will access this device function in parallel 
*/
__device__ bool compare_token(const char* data, const char* token, int token_len) {
    // loop around the token length to check if the current index is a match 
    for (int i = 0; i < token_len; ++i) {
        if (data[i] != token[i])
            return false;
    }
    return true;
}

/*
 GPU kernel that performs the count for word occurences 
 __global__ keyword used to declare the function as a kernel
*/
__global__ void count_occurrences_gpu(const char* data, int data_len, const char* token, int token_len, int* result) {
    // calculate the unique index to allow all threads to work in parallel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check to see if the index is in valid bounds before beginning 
    if (idx + token_len > data_len) return;

    // calls the device function to compare the token and see if that match is a non letter 
    if (compare_token(&data[idx], token, token_len)) {
        bool prefix_valid = (idx == 0 || is_non_letter(data[idx - 1]));
        bool suffix_valid = (idx + token_len == data_len || is_non_letter(data[idx + token_len]));

        // if the token match is a valid match then use atomic add to increment the counter 
        if (prefix_valid && suffix_valid) {
            atomicAdd(result, 1);
        }
    }
}

/*
 Modified function that now performs CUDA operations such as memory allocations and calling a CUDA kernel
*/
int calc_token_occurrences_gpu(const std::vector<char>& data, const char* token) {
    // Record the length of data and token as these values are needed for block calculation 
    int data_len = data.size();
    int token_len = strlen(token);

    // Allocate device memory
    char* d_data;
    char* d_token;
    int* d_result;
    int h_result = 0;

    // Use hipMalloc to allocate memory on the device for the incoming data
    hipMalloc(&d_data, data_len * sizeof(char));
    hipMalloc(&d_token, token_len * sizeof(char));
    hipMalloc(&d_result, sizeof(int));

    // Copy data to device, note that HostToDevice is used 
    hipMemcpy(d_data, data.data(), data_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_token, token, token_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Define block and thread sizes
    int numThreads = 256;
    // formula is used to dynamically calculate the block size to ensure the whole incoming file can be searched
    int numBlocks = (data_len + numThreads - 1) / numThreads;

    // Launch kernel
    count_occurrences_gpu<<<numBlocks, numThreads>>>(d_data, data_len, d_token, token_len, d_result);

    // Copy result back to host
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
    hipFree(d_token);
    hipFree(d_result);

    return h_result;
}

/*
 Original C++ function
 No changes in this build 
*/
int calc_token_occurrences(const std::vector<char>& data, const char* token)
{
    int numOccurrences = 0;
    int tokenLen = int(strlen(token));
    for (int i = 0; i< int(data.size()); ++i)
    {
        // test 1: does this match the token?
        auto diff = strncmp(&data[i], token, tokenLen);
        if (diff != 0)
            continue;

        // test 2: is the prefix a non-letter character?
        auto iPrefix = i - 1;
        if (iPrefix >= 0 && data[iPrefix] >= 'a' && data[iPrefix] <= 'z')
            continue;

        // test 3: is the prefix a non-letter character?
        auto iSuffix = i + tokenLen;
        if (iSuffix < int(data.size()) && data[iSuffix] >= 'a' && data[iSuffix] <= 'z')
            continue;
        ++numOccurrences;
    }
    return numOccurrences;
}
/*
 Original C++ function
 No changes in this build
*/
std::vector<char> read_file(const char* filename) {
    std::ifstream file(filename, std::ios::binary);

    if (!file) {
        std::cerr << "Error: Could not open the file " << filename << std::endl;
        return {};
    }

    file.seekg(0, std::ios::end);
    std::streamsize fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    std::vector<char> buffer(fileSize);

    if (!file.read(buffer.data(), fileSize)) {
        std::cerr << "Error: Could not read the file content." << std::endl;
        return {};
    }

    file.close();

    std::transform(buffer.begin(), buffer.end(), buffer.begin(), [](char c) { return std::tolower(c); });

    return buffer;
}

/*
 C++ function that calculates the current device specs 
*/
int deviceInformation() {

    // Get number of devices on system
    int deviceCount;
    gpuErrchk(hipGetDeviceCount(&deviceCount));

    std::cout << "Number of devices: " << deviceCount << std::endl;
    for (int i = 0; i < deviceCount; ++i)
    {
        // Get properties for device
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device " << i << std::endl;
        std::cout << "Name " << deviceProp.name << std::endl;
        std::cout << "Revision " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Memory " << deviceProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "Warp Size " << deviceProp.warpSize << std::endl;
        std::cout << "Clock " << deviceProp.clockRate << std::endl;
        std::cout << "Multiprocessors " << deviceProp.multiProcessorCount << std::endl;
    }
    return 0;
}

/*
 Main function that has been modified to allow the user to select which data file they wish to search through
 CPU and GPU comparsions are done through performace tracking tools 
 The record times are compared and shown to the user for added clarity 
 Option is given to the user to restart the application for an additional search
*/
int main() {

    // Display the current machines hardware 
    deviceInformation();
    
    // List of words to use as search tokens alongside file and file choice variables 
    const char* words[] = { "sword", "fire", "death", "love", "hate", "the", "man", "woman" };
    const char* filepath; 
    int fileChoice;
    int choice;

    // List of console messages to list the options to the user 
    std::cout << "\nWelcome to the application! Please enter the number of the text file to search: " << std::endl;
    std::cout << "1. shakespeare " << std::endl;
    std::cout << "2. beowulf " << std::endl;
    std::cout << "3. crime and punishment " << std::endl;
    std::cout << "4. edgar allan poe " << std::endl;
    std::cout << "5. pride and prejudice \n" << std::endl;

    // Users choice is stored 
    std::cin >> fileChoice;

    // Switch statement to handle the users choice and store the chosen file path
    switch(fileChoice) {
    case 1:
        filepath = "dataset/shakespeare.txt";
        std::cout << "Now searching shakespeare.txt for word occurences" << std::endl;
        break;
    case 2: 
        filepath = "dataset/beowulf.txt";
        std::cout << "Now searching beowulf.txt for word occurences" << std::endl;
        break;
    case 3:
        filepath = "dataset/crime_and_punishment.txt";
        std::cout << "Now searching crime_and_punishment.txt for word occurences" << std::endl;
        break;
    case 4:
        filepath = "dataset/edgar_allan_poe.txt";
        std::cout << "Now searching edgar_allan_poe.txt for word occurences" << std::endl;
        break;
    case 5:
        filepath = "dataset/pride_and_prejudice.txt";
        std::cout << "Now searching pride_and_prejudice.txt for word occurences" << std::endl;
        break;
    default:
        std::cout << "You must select a valid option!\n" << std::endl;
        main();
    }

    // the chosen file path is transfered to the read file function to open
    std::vector<char> file_data = read_file(filepath);
    if (file_data.empty()) return -1;

    // CPU version timing using chrono libary
    std::cout << "\nCPU version of text search:" << std::endl;
    auto cpu_start = std::chrono::high_resolution_clock::now();
    for (const char* word : words) {
        int occurrences = calc_token_occurrences(file_data, word);
        std::cout << "Found " << occurrences << " occurrences of word: " << word << std::endl;
    }
    auto cpu_end = std::chrono::high_resolution_clock::now();
    int cpu_duration = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end - cpu_start).count();
    std::cout << "CPU version took " << cpu_duration << "ms." << std::endl;


    // GPU version timing
    std::cout << "\nGPU version of text search:" << std::endl;
    hipEvent_t gpu_start, gpu_stop;
    float gpu_time = 0.0f;

    // cuda Events used to create two events to track the start and end timings 
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    // recording starts to track the timing 
    hipEventRecord(gpu_start);

    for (const char* word : words) {
        int occurrences = calc_token_occurrences_gpu(file_data, word);
        std::cout << "Found " << occurrences << " occurrences of word: " << word << std::endl;
    }

    // Records the end timing and synchronizes to ensure all operations are completed before recording the time
    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    // Adds the two timings together 
    hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop);

    // Round GPU time to the nearest millisecond
    int gpu_duration = static_cast<int>(std::round(gpu_time));
    std::cout << "GPU version took " << gpu_duration << "ms." << std::endl;

    // Cleanup CUDA events
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);


    // Speed comparison
    if (gpu_duration < cpu_duration) {
        std::cout << "\nThe GPU version is faster by " << (cpu_duration - gpu_duration) << "ms." << std::endl;
    }
    else if (cpu_duration < gpu_duration) {
        std::cout << "\nThe CPU version is faster by " << (gpu_duration - cpu_duration) << "ms." << std::endl;
    }
    else {
        std::cout << "\nBoth versions took the same amount of time." << std::endl;
    }

    // User is giving the option to restart the program 
    std::cout << "\nTo select another file enter 1 or to exit enter 0" << std::endl;
    std::cin >> choice;

    if (choice == 1)
    {
        main();
    }
    else
    {
        return 0;
    }

  
   
}








