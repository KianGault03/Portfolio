#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpuErrchk.h"

#include <stdio.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <>

constexpr size_t ELEMENTS = 2048;

__global__ void myKernal(const int* A, const int* B, int* C) {

    unsigned int block_idx = blockIdx.x;
    unsigned int thread_idx = threadIdx.x;
    unsigned int block_dim = blockDim.x;
    unsigned int idx = (block_idx * block_dim) + thread_idx;

    C[idx] = A[idx] + B[idx];

}

int Addition() {

    auto data_size = sizeof(int) * ELEMENTS;
    std::vector<int> A(ELEMENTS);
    std::vector<int> B(ELEMENTS);
    std::vector<int> C(ELEMENTS);

    for (unsigned int i = 0; i < ELEMENTS; i++)
    {
        A[i] = B[i] = i;
    }

    int* buffer_A, * buffer_B, * buffer_C;

    hipMalloc((void**)&buffer_A, data_size);
    hipMalloc((void**)&buffer_B, data_size);
    hipMalloc((void**)&buffer_C, data_size);

    hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);
    hipMemcpy(buffer_A, &B[0], data_size, hipMemcpyHostToDevice);

    myKernal<<<ELEMENTS / 1024, 1024 >>>(buffer_A, buffer_B, buffer_C);

    hipDeviceSynchronize();

    hipMemcpy(&C[0], buffer_C, data_size, hipMemcpyDeviceToHost);

    hipFree(buffer_A);
    hipFree(buffer_B);
    hipFree(buffer_C);

    for (int i = 0; i < 2048; ++i)
        if (C[i] != i + i)
            std::cout << "Error: " << i << std::endl;

    std::cout << "Finished" << std::endl;

    return 0;
}